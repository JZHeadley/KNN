#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : KNN.cu
 Author      : jzheadley
 Version     :
 Copyright   :
 Description : CUDA compute reciprocals
 ============================================================================
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <stdint.h>
 #include <float.h>
 #include <math.h>
 #include <iostream>
 #include <limits.h>
 
 #include "libarff/arff_parser.h"
 
 #include "knn-testing.h"

 using namespace std;
 #define K 3
 
 #define NUM_STREAMS 2
 
 __global__ void computeDistances(int numInstances, int numAttributes, float* dataset, float* distances)
 {
     int tid = blockDim.x * blockIdx.x + threadIdx.x;
     int row = tid / numInstances; // instance1Index
     int column = tid - ((tid / numInstances) * numInstances); //instance2Index
     if ((tid < numInstances * numInstances))
     {
         float sum = 0;
         int instance1 = row * numAttributes;
         int instance2 = column * numAttributes;
         for (int atIdx = 0; atIdx < numAttributes - 1; atIdx++) // numAttributes -1 since we don't want to compare class in the distance because that doesn't make sense
         {
             sum += ((dataset[instance1 + atIdx] - dataset[instance2 + atIdx]) * (dataset[instance1 + atIdx] - dataset[instance2 + atIdx]));
         }
         distances[row * numInstances + column] = (float) sqrt(sum);
         distances[column * numInstances + row] = distances[row * numInstances + column]; //set the distance for the other half of the pair we just computed
     }
 }
 
 __inline__ __device__ void reduceToK(float* distancesTo, int* indexes, int k, int curSize)
 {
     // we're just going to do a simple bubble sort and pretend the elements past k don't exist 
     // bubble sort because at this small of numbers it doesn't matter.  Granted K could be quite large and this would get costly, 
     // but you know... just don't do that to me I don't want to implement quicksort or really anything similar on a gpu
     float tmp;
     unsigned char idx;
     for (int i = 0; i < curSize - 1; i++)
     {
         for (int j = 0; j < curSize - i - 1; j++)
         {
             if (distancesTo[j] > distancesTo[j + 1])
             {
                 tmp = distancesTo[j];
                 idx = indexes[j];
                 distancesTo[j] = distancesTo[j + 1];
                 indexes[j] = indexes[j + 1];
                 distancesTo[j + 1] = tmp;
                 indexes[j + 1] = idx;
             }
         }
     }
 }
 __inline__ __device__ int vote(float* distancesTo, int *indexes, float* dataset, int k, int numAttributes)
 {
     int classVotes[32]; // can technically parallelize this reading in the class num and probably should come back and do that
     bool duplicate = false;
     int finalClass;
     int mostVotes = 0;
     for (int i = 0; i < 32; i++)
         classVotes[i] = 0;
     for (int i = 0; i < k; i++)
     {
         int classNum = dataset[indexes[i] * numAttributes + numAttributes - 1];
         classVotes[classNum] += 1;
     }
     for (int i = 0; i < 32; i++) // have to find highest count first
     {
         if (classVotes[i] > mostVotes)
         {
             finalClass = i;
             mostVotes = classVotes[i];
         }
     }
     for (int i = 0; i < 32; i++) // then compare to that to ensure we don't have duplicates
     {
         if (classVotes[i] == mostVotes && classVotes[i] > 0)
             duplicate = true;
     }
     if (duplicate)
     {
         if ((k - 1) > 0) // I'm not quite sure why I'm detecting dupes when k=1 but I am soo... this takes care of that and makes everything correct again...
             return vote(distancesTo, indexes, dataset, k - 1, numAttributes);
     }
     return finalClass;
 }
 
 __global__ void knn(int* predictions, float*distances, float*dataset, int numAttributes)
 {
     __shared__ int indexes[256];
     __shared__ float distancesTo[256];
     // gridDim.x is numInstances
     int bestInstanceId;
     float bestDistance = INT_MAX;
     int instanceFrom = blockIdx.x * gridDim.x;
     int distancePos;
     int rowBoundary = instanceFrom + gridDim.x - 1;
     if (blockDim.x < gridDim.x)
     { //If we have more elements than threads we need to do an inital reduction to fit into our shared mem
         if (threadIdx.x < blockDim.x) // only want 256 threads to come into this otherwise we will go out of bounds of our shared mem
         {
             for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) // will try to make this more coalesced later
             {
                 if (i == blockIdx.x) // don't need to include the diagonal
                     continue;
 
                 distancePos = instanceFrom + i;
                 if (distancePos > rowBoundary)
                 { // should take care of the final elements
                     break;
                 }
                 if (distances[distancePos] < bestDistance)
                 {
                     bestDistance = distances[distancePos];
                     bestInstanceId = i;
                 }
             }
             indexes[threadIdx.x] = bestInstanceId;
             distancesTo[threadIdx.x] = bestDistance;
         }
         __syncthreads();
 
         if (threadIdx.x < blockDim.x / 2) // only need the first half(128) of the threads to work on the 256 length shared mem arrays
         {
             int s;
             // this for should probably have the conditional of (s>>1) > k but if I do that I don't reduce enough sooo...
             // we're going with this until I find that error and just upping s back up after this for
             for (s = blockDim.x / 2; (s) > K; s >>= 1)
             {
                 if (threadIdx.x < s)
                 {
                     if (distancesTo[threadIdx.x + s] < distancesTo[threadIdx.x])
                     {
                         distancesTo[threadIdx.x] = distancesTo[threadIdx.x + s];
                         indexes[threadIdx.x] = indexes[threadIdx.x + s];
                     }
                     __syncthreads();
                 }
             }
             s *= 2;
             __syncthreads();
             if (s > K && threadIdx.x == 1)
             { // we need to reduce it just a little more
                 reduceToK(distancesTo, indexes, K, s);
             }
             __syncthreads();
             if (threadIdx.x == 1) // Should be able to multithread a piece of this voting method but I can't seem to make it not break without singlethreading this portion
                 predictions[blockIdx.x] = vote(distancesTo, indexes, dataset, K, numAttributes);
         }
     }
 }
 
 int main(int argc, char* argv[])
 {
     if (argc != 2)
     {
         if (argc != 3)
         {
             cout << "Usage: ./main datasets/datasetFile.arff" << endl;
             exit(0);
         }
     }
 
     ArffParser parser(argv[1]);
     ArffData* dataset = parser.parse();
 
    // I started with a streams first approach but then saw that the stream create takes forever so got rid of them to see if it mattered and 
    // it really didn't reduce the time either way... I suspect maybe if the number of instances is ridiculously large this might speed it up a 
    // little since I'm doing that copy async but theres probably not much reason for using streams aside from they're cool...
     hipStream_t *streams = (hipStream_t*) malloc(NUM_STREAMS * sizeof(hipStream_t));
     for (int i = 0; i < NUM_STREAMS; i++) // multiple streams
         hipStreamCreate(&streams[i]);
 
     int numInstances = dataset->num_instances();
     int numAttributes = dataset->num_attributes();
     printf("We're classifying %i instances with %i attributes\n", numInstances, numAttributes);
     hipEvent_t start, stop;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     float milliseconds = 0;
     int numTriangularSpaces = (numInstances * numInstances); //(numInstances * (numInstances - 1)) / 2; // don't actually need the diagonal since its all 0's so we can have numInstances-1 instead of + 1 but math is hard
     float* h_dataset, *h_distances;
     int*h_predictions;
     hipHostMalloc(&h_predictions, sizeof(int) * numInstances);
     hipHostMalloc(&h_dataset, sizeof(float) * numInstances * numAttributes);
     hipHostMalloc(&h_distances, sizeof(float) * numTriangularSpaces);
      printf("numTriangularSpaces is %i\n", numTriangularSpaces);
 
     for (int instanceNum = 0; instanceNum < numInstances; instanceNum++)
     {
         // each 'row' will be an instances
         // each 'column' a specific attribute
         ArffInstance* instance = dataset->get_instance(instanceNum);
         for (int attributeNum = 0; attributeNum < numAttributes; attributeNum++)
         {
             h_dataset[instanceNum * numAttributes + attributeNum] = (float) instance->get(attributeNum)->operator int32();
         }
 
     }
 
     float* d_dataset;
     float* d_distances;
     int* d_predictions;
 
     hipMalloc(&d_predictions, numInstances * sizeof(int));
     hipMalloc(&d_dataset, numInstances * numAttributes * sizeof(float));
     hipMalloc(&d_distances, numTriangularSpaces * sizeof(float));
 
     int threadsPerBlock = 512;
 //	int blocksPerGrid = (numInstances + threadsPerBlock - 1) / threadsPerBlock;
     int blocksPerGrid = ((numInstances * numInstances) + threadsPerBlock - 1) / threadsPerBlock;
     hipEventRecord(start);
 
     hipMemcpyAsync(d_dataset, h_dataset, numInstances * numAttributes * sizeof(float), hipMemcpyHostToDevice, streams[0]);
     hipMemcpyAsync(d_distances, h_distances, numTriangularSpaces * sizeof(float), hipMemcpyHostToDevice, streams[0]);
     computeDistances<<<blocksPerGrid, threadsPerBlock, 0, streams[0]>>>(numInstances, numAttributes, d_dataset, d_distances);
 
     hipMemcpyAsync(d_predictions, h_predictions, numInstances * sizeof(int), hipMemcpyHostToDevice, streams[1]);
     hipStreamSynchronize(streams[0]); // need this to ensure that the previous kernel computing the distances is finished otherwise we might not have the full distance matrix
     knn<<<numInstances, 256, 0, streams[1]>>>(d_predictions, d_distances, d_dataset, numAttributes);
     hipMemcpyAsync(h_predictions, d_predictions, numInstances * sizeof(int), hipMemcpyDeviceToHost, streams[1]);
 
     hipEventRecord(stop);
     hipEventSynchronize(stop);
     hipEventElapsedTime(&milliseconds, start, stop);
 
     hipError_t hipError_t = hipGetLastError();
 
     if (hipError_t != hipSuccess)
     {
         fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
         exit(EXIT_FAILURE);
     }
 
     int* confusionMatrix = computeConfusionMatrix(h_predictions, dataset);
     float accuracy = computeAccuracy(confusionMatrix, dataset);
 
     printf("The KNN classifier for %lu instances required %llu ms CPU time. Accuracy was %.4f\n", numInstances, (long long unsigned int) milliseconds,
             accuracy);
    hipHostFree(h_dataset);
    hipHostFree(h_distances);
    hipHostFree(h_predictions);
    hipFree(d_dataset);
    hipFree(d_distances);
    hipFree(d_predictions);
    return 0;
 }
 
